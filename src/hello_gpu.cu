
#include <hip/hip_runtime.h>
#include <stdio.h>
 
 
__global__ void helloGPU( char *str ) {
 
  printf( "Hello %s from GPU thread %d in block %d\n", str, threadIdx.x, blockIdx.x );
 
}
 
 
int main(int argc, char *argv[]) {

  const int N = 8;
  const int csize = N*sizeof(char);
  const int no_blocks = 4;
  const int no_threads = 5;
  char str[8];
  char *gstr;

  scanf("%s", str);

  hipMalloc( (void**)&gstr, csize );
  hipMemcpy( gstr, str, csize, hipMemcpyHostToDevice );

  helloGPU<<<no_blocks,no_threads>>>( gstr ); 
  hipDeviceSynchronize();
 
  hipFree( gstr );
  return 0;
}
